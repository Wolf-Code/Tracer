#include "hip/hip_runtime.h"
#include "CUDAIncluder.h"
#include "Raytracer.h"

extern "C"
{
    __device__ float clamp( float, float, float );

    __constant__ Object* ObjectArray;
    __constant__ unsigned int Objects;
    __constant__ CamData Camera;
    __constant__ long Seed;
    /*
    __global__ void TraceKernel( float3* Input, float3* Output )
    {
        //      Which block # of T in B      ID of Thread
        int x = ( blockIdx.x * blockDim.x ) + threadIdx.x;
        int y = ( blockIdx.y * blockDim.y ) + threadIdx.y;

        if ( x < Camera.Width && y < Camera.Height )
        {
            int ID = y * ( int )Camera.Width + x;

            hiprandState RandState;
            hiprand_init( Seed + ID, 0, 0, &RandState );

            Ray R = Camera.GetRay( x, y );

            Output[ ID ] = Input[ ID ] + Radiance<0>( &R, ObjectArray, Objects, &RandState );
        }
    }
    */

    __device__ float clamp( float X, float Min, float Max )
    {
        return fmaxf( Min, fminf( X, Max ) );
    }

    __global__ void TraceKernelRegion( float3* Input, int StartX, int StartY, int EndX, int EndY, float3* Output )
    {
        //      Which block # of T in B      ID of Thread
        int x = StartX + ( blockIdx.x * blockDim.x ) + threadIdx.x;
        int y = StartY + ( blockIdx.y * blockDim.y ) + threadIdx.y;

        if ( x < EndX && y < EndY )
        {
            int ID = y * ( int )Camera.Width + x;

            hiprandState RandState;
            hiprand_init( Seed + ID, 0, 0, &RandState );
            
            float JitteredX = clamp( x + ( hiprand_uniform( &RandState ) * 2.0f - 1.0f ) * 0.5f, 0, Camera.Width );
            float JitteredY = clamp( y + ( hiprand_uniform( &RandState ) * 2.0f - 1.0f ) * 0.5f, 0, Camera.Height );

            Ray R = Camera.GetRay( JitteredX, JitteredY );

            Output[ ID ] = Input[ ID ] + Radiance<0>( &R, ObjectArray, Objects, &RandState );
        }
    }
}

int main( )
{
    return 0;
}